
#include "common.h"
#include "timer.h"

void grav_cpu(float m0, float* m, float* r, float* F, int numBodies) {
    for(unsigned int i = 0; i < numBodies; ++i) {
        F[i] = G*m0*m[i]/(r[i]*r[i]);
    }
}

int main(int argc, char**argv) {

    hipDeviceSynchronize();

    // Allocate memory and initialize data
    Timer timer;
    float m0 = rand();
    unsigned int numBodies = (argc > 1)?(atoi(argv[1])):1048576;
    float* m = (float*) malloc(numBodies*sizeof(float));
    float* r = (float*) malloc(numBodies*sizeof(float));
    float* F_cpu = (float*) malloc(numBodies*sizeof(float));
    float* F_gpu = (float*) malloc(numBodies*sizeof(float));
    for (unsigned int i = 0; i < numBodies; ++i) {
        m[i] = rand();
        r[i] = rand();
    }

    // Compute on CPU
    startTime(&timer);
    grav_cpu(m0, m, r, F_cpu, numBodies);
    stopTime(&timer);
    printElapsedTime(timer, "CPU time");

    // Compute on GPU
    startTime(&timer);
    grav_gpu(m0, m, r, F_gpu, numBodies);
    stopTime(&timer);
    printElapsedTime(timer, "GPU time");

    // Verify result
    for(unsigned int i = 0; i < numBodies; ++i) {
        float diff = (F_cpu[i] - F_gpu[i])/F_cpu[i];
        const float tolerance = 0.00001;
        if(diff > tolerance || diff < -tolerance) {
            printf("Mismatch at index %u (CPU result = %e, GPU result = %e)\n", i, F_cpu[i], F_gpu[i]);
            exit(0);
        }
    }

    // Free memory
    free(m);
    free(r);
    free(F_cpu);
    free(F_gpu);

    return 0;

}

