#include "hip/hip_runtime.h"

#include "common.h"
#include "timer.h"

__global__ void grav_kernel(float m0, float* m, float* r, float* F, int numBodies) {

    // TODO
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if(i<numBodies){
		F[i] = G*m0*m[i]/(r[i]*r[i]);
	}
		
    




}

void grav_gpu(float m0, float* m, float* r, float* F, int numBodies) {

    Timer timer;

    // Allocate GPU memory
    startTime(&timer);

    // TODO
	float *m_d, *r_d, *F_d;
	hipMalloc((void**) &m_d, numBodies*sizeof(float));
	hipMalloc((void**) &r_d, numBodies*sizeof(float));
	hipMalloc((void**) &F_d, numBodies*sizeof(float));





    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation time");

    // Copy data to GPU
    startTime(&timer);

    // TODO
	
	hipMemcpy(m_d, m, numBodies*sizeof(float), hipMemcpyHostToDevice); 
	hipMemcpy(r_d, r, numBodies*sizeof(float), hipMemcpyHostToDevice);



    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy to GPU time");

    // Call kernel
    startTime(&timer);

    // TODO

	grav_kernel(m0, m, r, F, numBodies);




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Kernel time");

    // Copy data from GPU
    startTime(&timer);

    // TODO
	hipMemcpy(F, F_d, numBodies*sizeof(float), hipMemcpyDeviceToHost);





    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");

    // Free GPU memory
    startTime(&timer);

    // TODO

	hipFree(m_d); 
	hipFree(r_d); 
	hipFree(F_d);




    hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Deallocation time");

}

